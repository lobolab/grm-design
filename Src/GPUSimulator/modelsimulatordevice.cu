#include "hip/hip_runtime.h"
// Copyright (c) Lobo Lab (lobolab.umbc.edu)
// All rights reserved.

#include "csimstate.cuh"
#include "modelsimulatordevice.cuh"
#include "cerrorhandle.h"
#include "cexperimentsdata.h"
#include "cmodeldata.h"
#include "csimtempdata.h"
#include "csimop.cuh"
#include "Common/log.h"
#include <QElapsedTimer>

#include <>
#include <hip/device_functions.h>
#include <time.h>

namespace LoboLab {

// Storage in shared memory
struct CSharedData {
  double *tempHill;
  double *reductionArray;

  __device__ void initialize() {
    extern __shared__ double shared_mem[];
    tempHill = shared_mem; // nThreads 
    reductionArray = &shared_mem[blockDim.x]; // nThreads
  }
};


// Storage in local thread memory
struct CThreadData {
  int startIndex;
  int endIndex;
  int dimX;
  int dimY;

  __device__ void initialize(CExperimentsData *cExpData) {
    dimX = cExpData->inSimStates[0]->width();
    dimY = cExpData->inSimStates[0]->height();
    int nCellsPerThread = dimX * dimY / blockDim.x;
    int col = threadIdx.x;
    startIndex = col * nCellsPerThread;
    endIndex = startIndex + nCellsPerThread;
  }

};


template <class T>
__device__ inline const T& min(const T& a, const T& b) {
  return (!(b < a)) ? a : b;
}

template <class T>
__device__ inline const T& max(const T& a, const T& b) {
  return (a < b) ? b : a;
}

__device__ void computeOps(CSimOp* ops, int nOps) {
  for (int i = 0; i < nOps; ++i)
    ops[i].compute();   
}


__device__ void computeRatios(CModelData *cModelData, CThreadData *cThreadData, 
    CSimTempData *cSimTempData, int row, int col, CSimState *simState) {
  for (int i = 0; i < cModelData->nProducts; ++i)
    cSimTempData->ratios[blockDim.x*i + threadIdx.x] = 
      cModelData->limits[i] * cSimTempData->ratios[blockDim.x*i + threadIdx.x] 
      - cModelData->degradations[i] * cSimTempData->oldConcs[blockDim.x*i + threadIdx.x];
    
  for (int d = 0; d < cModelData->nDif; ++d){
    double diffusion = 0.0;
    double total = 0.0;
    int k = cModelData->difProdInd[d];

    if (row > 0){
      diffusion += simState->getProduct(row - 1, col,k);
      ++total;
    }
    if (row < cThreadData->dimX - 1){
      diffusion += simState->getProduct(row + 1, col, k);
      ++total;
    }
    if (col > 0){
      diffusion += simState->getProduct(row, col - 1, k);
      ++total;
    }
    if (col < cThreadData->dimY - 1){
      diffusion += simState->getProduct(row, col + 1, k);
      ++total;
    }
      
    if (diffusion > 0 || (total > 0 && cSimTempData->oldConcs[blockDim.x*k + threadIdx.x] > 0))
      cSimTempData->ratios[blockDim.x*k + threadIdx.x] += 
        cModelData->difConsts[d] * 
        (diffusion - total * 
          cSimTempData->oldConcs[blockDim.x*k + threadIdx.x]);
  }
}


__device__ double computeError(CExperimentsData *cExpData, CSharedData *cSharedData, 
    CThreadData *cThreadData, CSimState* simState, CSimState* targetSimState) {
  
  int lowerBound = -(cExpData->kernel / 2) + (cExpData->kernel % 2 == 0);  // Second part is to check if the kernel size is even or odd
  int upperBound = cExpData->kernel / 2;

  double dist = 0.0;
  // int kernelSize2 = cExpData->kernel * cExpData->kernel;
  for (int i = cThreadData->startIndex; i < cThreadData->endIndex; ++i) {
    int row = i / cThreadData->dimX;
    int col = i % cThreadData->dimX;
    for (int k = 0; k < cExpData->nTargetMorphogens; ++k) {
      double val1 = 0;
      double val2 = 0;
      // double val2 = targetSimState->getProduct(row, col, k);

      int count = 0;
      int minI = max(0, row + lowerBound);
      int maxI = min(cThreadData->dimX - 1, row + upperBound);
      int minJ = max(0, col + lowerBound);
      int maxJ = min(cThreadData->dimY - 1, col + upperBound);
      for (int ki = minI; ki <= maxI; ++ki) {
        for (int kj = minJ; kj <= maxJ; ++kj) {
            val1 += simState->getProduct(ki, kj, k + cExpData->nInputMorphogens);
            val2 += targetSimState->getProduct(ki, kj, k);
            count++;
        }
      }
      double absSub = (fabs(val1 - val2) / count) - cExpData->distErrorThreshold;

      if (absSub > 0)
        dist += log(1 + absSub);
      }
  }
    
  dist /= (cThreadData->dimX * cThreadData->dimY);

  cSharedData->reductionArray[threadIdx.x] = dist;

  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (threadIdx.x < s)
      cSharedData->reductionArray[threadIdx.x] = cSharedData->reductionArray[threadIdx.x] + 
                                                  cSharedData->reductionArray[threadIdx.x + s];
    __syncthreads();
  }

  dist = cSharedData->reductionArray[0];

  return dist;
}

  
__device__ void initializeSimState(CModelData *cModelData, CThreadData *cThreadData, 
    CSimTempData *cSimTempData, CSimState *inSimState) {
  int nInProducts = inSimState->nProducts(); // This is the number of input products (used for initialization of the state)
  int nProducts = cModelData->nProducts; // This is the number of products in the model, used during the simulation
  for (int i = cThreadData->startIndex; i < cThreadData->endIndex; ++i) {
    int row = i / cThreadData->dimX;
    int col = i % cThreadData->dimX;
    for (int k = 0; k < nInProducts; ++k) {
      double val1 = inSimState->getProduct(row, col, k);
      cSimTempData->cSimState1.setProduct(row, col, k, val1);
      cSimTempData->cSimState2.setProduct(row, col, k, val1); // Necessary because simulateExperiment only writes when conc changes
    }
    for (int k = nInProducts; k < nProducts; ++k) {
      cSimTempData->cSimState1.setProduct(row, col, k, 0.0);
      cSimTempData->cSimState2.setProduct(row, col, k, 0.0); // Necessary because simulateExperiment only writes when conc changes
    }
  }
}


__device__ double simulateExperiment(CExperimentsData *cExpData, CModelData *cModelData, 
    CSharedData *cSharedData, CThreadData *cThreadData, CSimTempData *cSimTempData, 
    CSimState *inSimState, CSimState **outSimState) {
  initializeSimState(cModelData, cThreadData, cSimTempData, inSimState);

  CSimState *simStateA = &cSimTempData->cSimState1;
  CSimState *simStateB = &cSimTempData->cSimState2;

  double change = 1.0;
  int step = 0;
  while (step < cExpData->nMaxSteps && change > cExpData->minConcChange) {
    double maxChange = 0.0;
    for (int i = cThreadData->startIndex; i < cThreadData->endIndex; ++i) {
      int row = i / cThreadData->dimX;
      int col = i % cThreadData->dimX;
      for (int k = 0; k < cModelData->nProducts; ++k)
        cSimTempData->oldConcs[blockDim.x*k + threadIdx.x] = simStateA->getProduct(row, col, k);

      computeOps(cModelData->ops, cModelData->nOps);
      computeRatios(cModelData, cThreadData, cSimTempData, row, col, simStateA);

      for (int k = 0; k < cModelData->nProducts; ++k) {
        // Skip constant products
        if (!(k < inSimState->nProducts() && inSimState->getProduct(row, col, k) > 0)) {
          double ratio = cSimTempData->ratios[blockDim.x * k + threadIdx.x];

          if (ratio) {
            double c = cSimTempData->oldConcs[blockDim.x * k + threadIdx.x] + cExpData->dt * ratio;
            if (c < cExpData->minConc)
              simStateB->setProduct(row, col, k, 0);
            else
              simStateB->setProduct(row, col, k, c);

            maxChange = fmax(maxChange, fabs(ratio));
          }
        }
      }
    }

    cSharedData->reductionArray[threadIdx.x] = maxChange;

    __syncthreads();
      
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
      if (threadIdx.x < s)
        cSharedData->reductionArray[threadIdx.x] = fmax(cSharedData->reductionArray[threadIdx.x], 
                                                        cSharedData->reductionArray[threadIdx.x + s]);
      __syncthreads();
    }

    change = cSharedData->reductionArray[0];

    CSimState *temp = simStateA;
    simStateA = simStateB;
    simStateB = temp;
      
    ++step;
  }

  *outSimState = simStateA; // simStateA can be simState1 or simState2

  return change;
}

 
__global__ void __launch_bounds__(NTHREADS, 1) simulateModelKernel(CExperimentsData *cExpData, 
    CModelData *cModelData, CSimTempData *cSimTempData, double *return_error, long long int *return_time) {

  long long int start;
  if (threadIdx.x == 0)
  {
    start = clock64();
  }

  CSharedData cSharedData;
  cSharedData.initialize();

  CThreadData cThreadData;
  cThreadData.initialize(cExpData);

  if (threadIdx.x == 0) {
    for (int i = 0; i < cModelData->nOps; ++i)
      cModelData->ops[i].linkFuncPointer(cSimTempData->ratios, cSimTempData->oldConcs, blockDim.x);
  }

  __syncthreads();
    
  int i = 0;
  double change = 0.0;
  double error = 0.0;
  while (i < cExpData->nExperiments && error <= cModelData->maxError) {
    CSimState *inSimState = cExpData->inSimStates[i];
    CSimState *outSimState;
    change += simulateExperiment(cExpData, cModelData, &cSharedData, &cThreadData, cSimTempData, inSimState, &outSimState);

    CSimState *targetSimState = cExpData->targetSimStates[i];
    double expError = computeError(cExpData, &cSharedData, &cThreadData, outSimState, targetSimState);

    if (change > cExpData->minConcChange)
      expError += 0.01 * (change - cExpData->minConcChange);// log(1 + ((change_val - minConcChange) / minConcChange));
      
    error += expError / cExpData->nExperiments;

    ++i;
  }

  long long int elapsedTime;
  if (threadIdx.x == 0) {
    elapsedTime = (clock64() - start);
    *return_time = elapsedTime;
  }

  *return_error = error;
}

//#include "stdlib.h"

void launchKernel(CExperimentsData *cExperimentsDataDevice, CModelData *cModelDataDevice, 
    CSimTempData *cSimTempDataDevice, int nProducts, hipStream_t stream, 
    double *return_error, long long int *return_time) {
  const int sharedMemorySize = sizeof(double) * (2 * NTHREADS);

  if (sharedMemorySize > MAXSHAREDMEM)
    Log::write() << "launchKernel: ERROR: Requesting " <<
      sharedMemorySize << " shared memory of " << MAXSHAREDMEM << " available. (nProducts = " << nProducts << ")" << endl;

  simulateModelKernel <<< 1, NTHREADS, sharedMemorySize, stream >>> (cExperimentsDataDevice, cModelDataDevice, cSimTempDataDevice, return_error, return_time);

}

}

